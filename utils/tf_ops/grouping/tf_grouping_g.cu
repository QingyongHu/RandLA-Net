
#include <hip/hip_runtime.h>
// input: radius (1), nsample (1), xyz1 (b,n,3), xyz2 (b,m,3)
// output: idx (b,m,nsample), pts_cnt (b,m)
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        int cnt = 0;
        for (int k=0;k<n;++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float x2=xyz2[j*3+0];
            float y2=xyz2[j*3+1];
            float z2=xyz2[j*3+2];
            float x1=xyz1[k*3+0];
            float y1=xyz1[k*3+1];
            float z1=xyz1[k*3+2];
    	    float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
            if (d<radius) {
                if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
                    for (int l=0;l<nsample;++l)
                        idx[j*nsample+l] = k;
                }
                idx[j*nsample+cnt] = k;
                cnt+=1;
            }
        }
        pts_cnt[j] = cnt;
    }
}

// input: points (b,n,c), idx (b,m,nsample)
// output: out (b,m,nsample,c)
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
    }
}

// input: grad_out (b,m,nsample,c), idx (b,m,nsample), 
// output: grad_points (b,n,c)
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m*nsample*batch_index;
    grad_out += m*nsample*c*batch_index;
    grad_points += n*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                 atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
            }
        }
    }
}

// input: k (1), distance matrix dist (b,m,n)
// output: idx (b,m,n), dist_out (b,m,n)
// only the top k results within n are useful
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist+=m*n*batch_index;
    outi+=m*n*batch_index;
    out+=m*n*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // copy from dist to dist_out
    for (int j=index;j<m;j+=stride) {
        for (int s=0;s<n;++s) {
            out[j*n+s] = dist[j*n+s];
            outi[j*n+s] = s;
        }
    }

    float *p_dist;
    for (int j=index;j<m;j+=stride) {
        p_dist = out+j*n;
        // selection sort for the first k elements
        for (int s=0;s<k;++s) {
            int min=s; 
            // find the min
            for (int t=s+1;t<n;++t) {
                if (p_dist[t]<p_dist[min]) {
                    min = t;
                }
            }
            // swap min-th and i-th element
            if (min!=s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j*n+min];
                outi[j*n+min] = outi[j*n+s];
                outi[j*n+s] = tmpi;
            }
        }
    }
}

void queryBallPointLauncher(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    query_ball_point_gpu<<<b,256>>>(b,n,m,radius,nsample,xyz1,xyz2,idx,pts_cnt);
    //cudaDeviceSynchronize();
}
void selectionSortLauncher(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    selection_sort_gpu<<<b,256>>>(b,n,m,k,dist,outi,out); 
    //cudaDeviceSynchronize();
}
void groupPointLauncher(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out){
    group_point_gpu<<<b,256>>>(b,n,c,m,nsample,points,idx,out);
    //cudaDeviceSynchronize();
}
void groupPointGradLauncher(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points){
    group_point_grad_gpu<<<b,256>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //group_point_grad_gpu<<<1,1>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //cudaDeviceSynchronize();
}
