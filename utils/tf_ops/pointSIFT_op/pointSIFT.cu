
#include <hip/hip_runtime.h>
__global__ void selection_k_radius_gpu(int b, int m, int k, float radius, const int* idx, const float* val, int* idx_out, float* val_out){
    int batch_index = blockIdx.x;
    int stride = batch_index * m * k;
    idx += stride;
    val += stride;
    idx_out += stride;
    val_out += stride;
    for(int i = threadIdx.x; i < m;i += blockDim.x) {

        for(int j = 0;j < k;j ++) {
            if(val[i * k + j] < radius) {
                idx_out[i * k + j] = idx[i * k + j];
                val_out[i * k + j] = val[i * k + j];
            } else {
                idx_out[i * k + j] = idx[i * k ];
                val_out[i * k + j] = val[i * k ];
            }
        }
    }
}

__global__ void cube_select(int b, int n,float radius, const float* xyz, int* idx_out) {
    int batch_idx = blockIdx.x;
    xyz += batch_idx * n * 3;
    idx_out += batch_idx * n * 8;
    float temp_dist[8];
    float judge_dist = radius * radius;
    for(int i = threadIdx.x; i < n;i += blockDim.x) {
        float x = xyz[i * 3];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        for(int j = 0;j < 8;j ++) {
            temp_dist[j] = 1e8;
            idx_out[i * 8 + j] = i; // if not found, just return itself..
        }
        for(int j = 0;j < n;j ++) {
            if(i == j) continue;
            float tx = xyz[j * 3];
            float ty = xyz[j * 3 + 1];
            float tz = xyz[j * 3 + 2];
            float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
            if(dist > judge_dist) continue;
            int _x = (tx > x);
            int _y = (ty > y);
            int _z = (tz > z);
            int temp_idx = _x * 4 + _y * 2 + _z;
            if(dist < temp_dist[temp_idx]) {
                idx_out[i * 8 + temp_idx] = j;
                temp_dist[temp_idx] = dist;
            }
        }
    }
}

__global__ void cube_select_two(int b, int n,float radius, const float* xyz, int* idx_out) {
    int batch_idx = blockIdx.x;
    xyz += batch_idx * n * 3;
    idx_out += batch_idx * n * 16;
    float temp_dist[16];
    float judge_dist = radius * radius;
    for(int i = threadIdx.x; i < n;i += blockDim.x) {
        float x = xyz[i * 3];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        for(int j = 0;j < 16;j ++) {
            temp_dist[j] = judge_dist;
            idx_out[i * 16 + j] = i; // if not found, just return itself..
        }
        for(int j = 0;j < n;j ++) {
            if(i == j) continue;
            float tx = xyz[j * 3];
            float ty = xyz[j * 3 + 1];
            float tz = xyz[j * 3 + 2];
            float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
            if(dist > judge_dist) continue;
            int _x = (tx > x);
            int _y = (ty > y);
            int _z = (tz > z);
            int temp_idx = _x * 8 + _y * 4 + _z * 2;
            bool flag = false;
            for(int k = 0;k < 2;k ++) {
                if (dist < temp_dist[temp_idx + k]) {
                    flag = true;
                }
                if (flag) {
                    for (int kk = 1; kk >= k + 1; kk --) {
                        idx_out[i * 16 + temp_idx + kk] = idx_out[i * 16 + temp_idx + kk - 1];
                        temp_dist[temp_idx + kk] = temp_dist[temp_idx + kk - 1];
                    }
                    idx_out[i * 16 + temp_idx + k] = j;
                    temp_dist[temp_idx + k] = dist;
                    break;
                }
            }

        }
    }
}

__global__ void cube_select_four(int b, int n,float radius, const float* xyz, int* idx_out) {
    int batch_idx = blockIdx.x;
    xyz += batch_idx * n * 3;
    idx_out += batch_idx * n * 32;
    float temp_dist[32];
    float judge_dist = radius * radius;
    for(int i = threadIdx.x; i < n;i += blockDim.x) {
        float x = xyz[i * 3];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        for(int j = 0;j < 32;j ++) {
            temp_dist[j] = judge_dist;
            idx_out[i * 32 + j] = i; // if not found, just return itself..
        }
        for(int j = 0;j < n;j ++) {
            if(i == j) continue;
            float tx = xyz[j * 3];
            float ty = xyz[j * 3 + 1];
            float tz = xyz[j * 3 + 2];
            float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
            if(dist > judge_dist) continue;
            int _x = (tx > x);
            int _y = (ty > y);
            int _z = (tz > z);
            int temp_idx = _x * 16 + _y * 8 + _z * 4;
            bool flag = false;
            for(int k = 0;k < 4;k ++) {
                if (dist < temp_dist[temp_idx + k]) {
                    flag = true;
                }
                if (flag) {
                    for (int kk = 3; kk >= k + 1; kk --) {
                        idx_out[i * 32 + temp_idx + kk] = idx_out[i * 32 + temp_idx + kk - 1];
                        temp_dist[temp_idx + kk] = temp_dist[temp_idx + kk - 1];
                    }
                    idx_out[i * 32 + temp_idx + k] = j;
                    temp_dist[temp_idx + k] = dist;
                    break;
                }
            }

        }
    }
}



void selectionKRadiusLauncher(int b, int m, int k, float radius, const int* idx, const float* val, int* idx_out, float* val_out){
    selection_k_radius_gpu<<<b,256>>>(b, m, k, radius, idx, val, idx_out, val_out);
}
void cubeSelectLauncher(int b, int n, float radius, const float* xyz, int* idx_out) {
    cube_select<<<b, 512>>>(b, n, radius, xyz, idx_out);
}
void cubeSelectTwoLauncher(int b, int n, float radius, const float* xyz, int* idx_out) {
    cube_select_two<<<b, 512>>>(b, n, radius, xyz, idx_out);
}
void cubeSelectFourLauncher(int b, int n, float radius, const float* xyz, int* idx_out) {
    cube_select_four<<<b, 512>>>(b, n, radius, xyz, idx_out);
}
